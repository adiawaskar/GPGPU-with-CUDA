#include <iostream>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
using namespace std;

const int N = 2;

__global__ void matAdd3d(int *a, int *b, int *c, int N ){
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int layer = blockIdx.z*blockDim.z + threadIdx.z;

    if(row < N && col < N && layer < N){
        c[layer*N*N + row*N + col] = a[layer*N*N + row*N + col] + b[layer*N*N + row*N + col];
    }
    __syncthreads();
}

void init_matrix(int *m, int N){
    for(int i = 0; i < N; i++){
        for(int j = 0; j<N; j++){
            for(int k = 0; k<N; k++){
                m[i*N*N + j*N + k]= 1;
            }
        }
    }
}

int main(){
    int size = N*N*N*sizeof(int);

    int *h_a = new int[N*N*N];
    int *h_b = new int[N*N*N];
    int *h_c = new int[N*N*N];

    init_matrix(h_a, N);
    init_matrix(h_b, N);

    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);
    hipDeviceSynchronize();

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    //allocating memory in kernel
    dim3 THREADS(N, N, N);
    dim3 BLOCKS(N, N, N);

    matAdd3d<<< BLOCKS, THREADS>>> (d_a, d_b, d_c, N);
    hipDeviceSynchronize();

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    cout<<"Matrix A: "<<endl;
    for(int i = 0; i < N; i++){
        for(int j = 0; j<N; j++){
            for(int k = 0; k<N; k++){
                cout<<h_a[i*N*N + j*N + k]<<" ";
            }
            cout<<endl;
        }
        cout<<endl;
    } 
    cout<<"Matrix B: "<<endl;
    for(int i = 0; i < N; i++){
        for(int j = 0; j<N; j++){
            for(int k = 0; k<N; k++){
                cout<<h_b[i*N*N + j*N + k]<<" ";
            }
            cout<<endl;
        }
        cout<<endl;
    }
    cout<<"Result:"<<endl;
    for(int i = 0; i < N; i++){
        for(int j = 0; j<N; j++){
            for(int k = 0; k<N; k++){
                cout<<h_c[i*N*N + j*N + k]<<" ";
            }
            cout<<endl;
        }
        cout<<endl;
    }
    //cout<<size<<" "<<threads<<" "<<blocks;
}