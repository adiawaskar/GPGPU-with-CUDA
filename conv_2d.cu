#include <iostream>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

using namespace std;

#define MASK_DIM 3

#define MASK_OFFSET (MASK_DIM/2)

__constant__ int mask[3*3];

__global__ void conv_2d(int *matrix, int *result, int N){
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;

    int start_r = row - MASK_OFFSET;
    int start_c = col - MASK_OFFSET;

    if(row<N && col<N){
        int temp = 0;
        //Iterate over all rows
    for(int i = 0; i<MASK_DIM; i++){
        //Go over each column
        for (int j = 0; j<MASK_DIM; j++){

            if(((start_r + i)>=0)&&((start_r + i)<N)){

                if(((start_c + j)>=0)&&((start_c + j)<N)){

                    temp += matrix[(start_r + i) * N + (start_c + j)] * mask[i * MASK_DIM + j];
                    
                }
            }
        }
    }

    result[row * N + col] = temp;
    }
}

void init_matrix(int *m, int N){

    //randomly generates a matrix
    for(int i = 0; i < N*N; i++){
        m[i] = 1;
    }
}

int main(){
    int N = 3;

    //initialise input and output matrix and allocate memory
    int size_n = N*N*sizeof(int);

    int *matrix = new int [N*N];
    int *result = new int [N*N];
    init_matrix(matrix, N);

    //initialise mask and allocate memory
    int size_m = MASK_DIM * MASK_DIM * sizeof(int);

    int *h_mask = new int [MASK_DIM * MASK_DIM];
    init_matrix(h_mask, MASK_DIM);
    
    //allocate device vectors and their memory
    int *d_matrix, *d_result;
    hipMalloc(&d_matrix, size_n);
    hipMalloc(&d_result, size_n);
    //cudaMalloc(&d_mask, size_m);

    hipMemcpy(d_matrix, matrix, size_n, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(mask), h_mask, size_m);

    int threads = 16;
    int blocks = (N + threads - 1)/ threads;

    dim3 block_dim(threads, threads);
    dim3 grid_dim(blocks, blocks);

    conv_2d<<< grid_dim, block_dim>>>(d_matrix, d_result, N );
    hipMemcpy(result, d_result, size_n, hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++){
        for(int j = 0; j<N; j++){
            cout<<matrix[i*N + j]<<" ";
        }
        cout<<endl;
    }
    cout<<endl;
    for(int i = 0; i < N; i++){
        for(int j = 0; j<N; j++){
            cout<<h_mask[i*N + j]<<" ";
        }
        cout<<endl;
    }
    cout<<endl;
    for(int i = 0; i < N; i++){
        for(int j = 0; j<N; j++){
            cout<<result[i*N + j]<<" ";
        }
        cout<<endl;
    }

    return 0;

}