#include <iostream>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

using namespace std;

//initialization
const  int  N = 2047;
const int blockSize = 1024;
 
const int gridSize = 4 ;

__global__ void minElem(float *image, int N, float *result){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    //double result;
    __shared__ float sdata[blockSize*sizeof(float)];

    if(id < N){
        //loads image values to shared data
        sdata[threadIdx.x] = image[id];
    }

    for(int s = (blockDim.x/2); s > 0; s >>= 1){

        if(threadIdx.x < s){
            //store the value of lower one in sdata[threadIdx.x]
            if(sdata[threadIdx.x] > sdata[threadIdx.x + s]){
                //if sdata[threadIdx.x + s] is lower store it in sdata[threadIdx.x]
                sdata[threadIdx.x] = sdata[threadIdx.x + s];
                //else let sdata[threadIdx.x] be as it is
            }
        }
    }
    //when all comparisons made load the value of lowest element in result and return in
    if(threadIdx.x == 0){
        result[blockIdx.x] = sdata[0];
    }
    //return result;
}
__global__ void vecSum(float *result, float *sum, int n){
    __shared__ float sdata[gridSize*sizeof(float)]; 
    /* SHARED MEMORY SHOULD BE THE SIZE OF NUMBER OF THREADS IN A BLOCK WHICH FOR THIS KERNEL 
    WILL BE gridSize*sizeof(float) and blockDim.x variable will hold the value gridSize   */
 
 
    int id = blockIdx.x*blockDim.x+threadIdx.x;
 
 
    if(id<n){
        sdata[threadIdx.x] = result[id];
    }
    __syncthreads();
 
    for(unsigned int s=blockDim.x/2; s > 0; s >>= 1){
        if(threadIdx.x<s)
        {
            
            if(sdata[threadIdx.x] > sdata[threadIdx.x + s])
            {
                //if sdata[threadIdx.x + s] is lower store it in sdata[threadIdx.x]
            
                sdata[threadIdx.x] = sdata[threadIdx.x + s];
                //else let sdata[threadIdx.x] be as it is
            }
        }
        __syncthreads();
    }
 
    if(threadIdx.x == 0){
        sum[blockIdx.x] = sdata[0];
    }
 
 
    /* The above code is literally the code from previous kernel with array names changed and the 
    element wise multiplication removed */ 
}
int main(){

    float *h_c, *h_sum, *h_r;
    float *d_c, *d_sum, *d_r;
    size_t size = N*sizeof(float);

    //memory allocation
    //h_a = (float*)malloc(size);
    //h_b = (float*)malloc(size);
    h_c = (float*)malloc(size);
    h_sum = (float*)malloc(gridSize*sizeof(float));
    h_r = (float*)malloc(1*sizeof(float));

    //cudaMalloc (&d_a,size);
    //cudaMalloc (&d_b, size);
    hipMalloc (&d_c, size);
    hipMalloc (&d_sum, gridSize*sizeof(float));
    hipMalloc (&d_r, 1*sizeof(float));

    int i;
    for(i=0; i<N; i++){
        h_c[i] = (i+1);
    }

    hipMemcpy( d_c, h_c, size, hipMemcpyHostToDevice);
    //cudaMemcpy( d_b, h_b, size, cudaMemcpyHostToDevice);

    
    //kernel call
    minElem<<<gridSize, blockSize>>>( d_c, N, d_sum); 
    vecSum<<<1, gridSize>>>(d_sum, d_r, gridSize);

    hipMemcpy( h_sum, d_sum, gridSize*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy( h_c, d_c, size, hipMemcpyDeviceToHost);

    hipMemcpy( h_r, d_r, 1*sizeof(float), hipMemcpyDeviceToHost);

   
    // printing each partial sum after first reduction : Just for visualization
    cout<<"Minimum value :  "<<endl;
    for( i=0; i<gridSize; i++){
        cout<<h_sum[i]<<",";
    }
    cout<<endl;

    for(int i = 0; i < N; i++){
        cout<<h_c[i]<<" ";
    }
    cout<<endl;

    //print result
    cout<<"Final result after adding all partial sums : "<<h_r[0]<<endl;

    //free memory
    //cudaFree(d_a);
    //cudaFree(d_b);
    hipFree(d_c);
    //free(h_a);
    //free(h_b);
    free(h_c);
    return 0;
}