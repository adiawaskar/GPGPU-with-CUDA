#include <iostream>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void vecAdd(float* a, float* b, float* c, int n, float* result){
    int id = blockIdx.x*blockDim.x+threadIdx.x;

    //multiplication
    if(id<n){
        c[id] = a[id] * b[id];
    }
    __syncthreads();

    //addition
    __shared__ float sdata[1024*sizeof(float)];
    
    
    if(id<n){
        sdata[threadIdx.x] = c[id];
    }
    __syncthreads();

    for(unsigned int s=blockDim.x/2; s > 0; s >>= 1){
        if(threadIdx.x<s){
            sdata[threadIdx.x] += sdata[threadIdx.x + s];
        }
        __syncthreads();
    }

    if(threadIdx.x == 0){
        result[blockIdx.x] = sdata[0];
    }
}
int main(){
    //initialization
    int N = 1025;
    float *h_a, *h_b, *h_c, *h_sum;
    float *d_a, *d_b, *d_c, *d_sum;
    size_t size = N*sizeof(float);

    int blockSize, gridSize;
    blockSize = 1024;
    gridSize = (int)ceil((float)N/blockSize);

    //memory allocation
    h_a = (float*)malloc(size);
    h_b = (float*)malloc(size);
    h_c = (float*)malloc(size);
    h_sum = (float*)malloc(size);

    hipMalloc (&d_a,size);
    hipMalloc (&d_b, size);
    hipMalloc (&d_c, size);
    hipMalloc (&d_sum, size);

    int i;
    for(i=0; i<N; i++){
        h_a[i] = 1.0;
        h_b[i] = 1.0;
    }

    hipMemcpy( d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, size, hipMemcpyHostToDevice);

    
    //kernel call
    vecAdd<<<gridSize, blockSize>>>( d_a, d_b, d_c, N, d_sum);
    //vecAdd<<< 1, blockSize>>>( d_a, d_b, d_c, N, d_sum);

    hipMemcpy( h_c, d_c, size, hipMemcpyDeviceToHost);
    hipMemcpy( h_sum, d_sum, size, hipMemcpyDeviceToHost);

   
    //printing each intermediate vector
    for( i=0; i<N; i++){
        cout<<h_c[i]<<" ";
    }
    cout<<endl;

    //print result
    cout<<"Final result: "<<h_sum[0]<<endl;

    //free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);
    return 0;
}